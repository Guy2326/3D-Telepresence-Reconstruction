#include "hip/hip_runtime.h"
#include "CudaHandleError.h"
#include "Parameters.h"
#include <Windows.h>
#include <iostream>
#include "Timer.h"
#include "Vertex.h"
#include "Parameters.h"
#include "TsdfVolume.cuh"

namespace tsdf {
	float3 size;
	float3 center;
	float3 volumeSize;
	float3 offset;

	float* volume_device;
	UINT8* volumeBin_device;
	UINT16* depth_device;
	uchar4* color_device;
	Transformation* depthTrans_device;
	Transformation* colorTrans_device;
	Intrinsics* depthIntrinsics_device;
	Intrinsics* colorIntrinsics_device;
	Vertex* vertex_device;
	int* count_device;
}
using namespace tsdf;

#if VOLUME == 256
CUDA_CALLABLE_MEMBER __forceinline__ int devicePid(int x, int y) {
	return (x & 15) | ((y & 15) << 4) | ((x >> 4) << 8) | ((y >> 4) << 12);
}

CUDA_CALLABLE_MEMBER __forceinline__ int deviceVid(int x, int y, int z) {
	return (x & 15) | ((y & 15) << 4) | ((z & 15) << 8) | ((x >> 4) << 12) | ((y >> 4) << 16) | ((z >> 4) << 20);
}
#elif VOLUME == 512
CUDA_CALLABLE_MEMBER __forceinline__ int devicePid(int x, int y) {
	return (x & 15) | ((y & 15) << 4) | ((x >> 4) << 8) | ((y >> 4) << 13);
}

CUDA_CALLABLE_MEMBER __forceinline__ int deviceVid(int x, int y, int z) {
	return (x & 15) | ((y & 15) << 4) | ((z & 15) << 8) | ((x >> 4) << 12) | ((y >> 4) << 17) | ((z >> 4) << 22);
}
#endif


extern "C"
void cudaInitVolume(float sizeX, float sizeY, float sizeZ, float centerX, float centerY, float centerZ) {
	size = make_float3(sizeX, sizeY, sizeZ);
	center = make_float3(centerX, centerY, centerZ);
	volumeSize = size * (1.0 / VOLUME);
	offset = center - size * 0.5;
	HANDLE_ERROR(hipMalloc(&volume_device, VOLUME * VOLUME * VOLUME * sizeof(float)));
	HANDLE_ERROR(hipMalloc(&volumeBin_device, VOLUME * VOLUME * VOLUME * sizeof(UINT8)));
	HANDLE_ERROR(hipMalloc(&depth_device, MAX_CAMERAS * DEPTH_H * DEPTH_W * sizeof(UINT16)));
	HANDLE_ERROR(hipMalloc(&color_device, MAX_CAMERAS * COLOR_H * COLOR_W * sizeof(uchar4)));
	HANDLE_ERROR(hipMalloc(&depthTrans_device, MAX_CAMERAS * sizeof(Transformation)));
	HANDLE_ERROR(hipMalloc(&colorTrans_device, MAX_CAMERAS * sizeof(Transformation)));
	HANDLE_ERROR(hipMalloc(&depthIntrinsics_device, MAX_CAMERAS * sizeof(Intrinsics)));
	HANDLE_ERROR(hipMalloc(&colorIntrinsics_device, MAX_CAMERAS * sizeof(Intrinsics)));
	HANDLE_ERROR(hipMalloc(&vertex_device, MAX_VERTEX * sizeof(Vertex)));
	HANDLE_ERROR(hipMalloc(&count_device, VOLUME * VOLUME * sizeof(int)));
}

extern "C"
void cudaReleaseVolume() {
	HANDLE_ERROR(hipFree(volume_device));
	HANDLE_ERROR(hipFree(volumeBin_device));
	HANDLE_ERROR(hipFree(depth_device));
	HANDLE_ERROR(hipFree(color_device));
	HANDLE_ERROR(hipFree(depthTrans_device));
	HANDLE_ERROR(hipFree(colorTrans_device));
	HANDLE_ERROR(hipFree(depthIntrinsics_device));
	HANDLE_ERROR(hipFree(colorIntrinsics_device));
	HANDLE_ERROR(hipFree(vertex_device));
	HANDLE_ERROR(hipFree(count_device));
}

__global__ void kernelIntegrateDepth(int cameras, float* volume, UINT8* volumeBin, Transformation* transformation, Intrinsics* intrinsics, float* depthMap, float3 volumeSize, float3 offset) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= VOLUME || y >= VOLUME) {
		return;
	}

	const float TRANC_DIST_M = 3.0 * max(volumeSize.x, max(volumeSize.y, volumeSize.z));

	struct VolumePara {
		float tsdf = 0;
		UINT8 cnt = 0;
		UINT8 bin = 0;
	} volumePara[VOLUME];

	for (int i = 0; i < cameras; i++) {
		float3 ori = make_float3(x, y, -1) * volumeSize + offset;
		float3 pos = transformation[i].translate(ori);
		float3 deltaZ = transformation[i].deltaZ() * volumeSize;

		for (int z = 0; z < VOLUME; z++) {
			float tsdf = -1;
			pos = pos + deltaZ;
			int2 pixel = intrinsics[i].translate(pos);

			if (pos.z > 0 && 0 <= pixel.x && pixel.x < DEPTH_W && 0 <= pixel.y && pixel.y < DEPTH_H) {
				UINT16 depth = depthMap[(i * DEPTH_H + pixel.y) * DEPTH_W + pixel.x];

				if (depth != 0) {
					float sdf = depth * 0.001 - pos.z;

					if (sdf >= -TRANC_DIST_M) {
						tsdf = sdf / TRANC_DIST_M;
					}
				}
			}

			if (tsdf != -1) {
				volumePara[z].tsdf += tsdf;
				volumePara[z].bin |= (1 << i);
				volumePara[z].cnt++;
			}
		}
	}

	for (int z = 0; z < VOLUME; z++) {
		int id = deviceVid(x, y, z);
		if (volumePara[z].bin != 0) {
			volume[id] = volumePara[z].tsdf / volumePara[z].cnt;
		} else {
			volume[id] = -1;
		}
		volumeBin[id] = volumePara[z].bin;
	}
}

__device__ __forceinline__ UINT16 deviceGetCubeIndex(float* volume, int x, int y, int z) {
	if (x + 1 >= VOLUME) return 0;
	if (y + 1 >= VOLUME) return 0;
	if (z + 1 >= VOLUME) return 0;
	if (volume[deviceVid(x + 0, y + 0, z + 0)] == -1) return 0;
	if (volume[deviceVid(x + 1, y + 0, z + 0)] == -1) return 0;
	if (volume[deviceVid(x + 0, y + 1, z + 0)] == -1) return 0;
	if (volume[deviceVid(x + 1, y + 1, z + 0)] == -1) return 0;
	if (volume[deviceVid(x + 0, y + 0, z + 1)] == -1) return 0;
	if (volume[deviceVid(x + 1, y + 0, z + 1)] == -1) return 0;
	if (volume[deviceVid(x + 0, y + 1, z + 1)] == -1) return 0;
	if (volume[deviceVid(x + 1, y + 1, z + 1)] == -1) return 0;
	UINT16 index = 0;
	if (volume[deviceVid(x + 0, y + 0, z + 0)] < 0) index |= 1;
	if (volume[deviceVid(x + 1, y + 0, z + 0)] < 0) index |= 2;
	if (volume[deviceVid(x + 0, y + 1, z + 0)] < 0) index |= 8;
	if (volume[deviceVid(x + 1, y + 1, z + 0)] < 0) index |= 4;
	if (volume[deviceVid(x + 0, y + 0, z + 1)] < 0) index |= 16;
	if (volume[deviceVid(x + 1, y + 0, z + 1)] < 0) index |= 32;
	if (volume[deviceVid(x + 0, y + 1, z + 1)] < 0) index |= 128;
	if (volume[deviceVid(x + 1, y + 1, z + 1)] < 0) index |= 64;
	return index;
}

__global__ void kernelMarchingCubesCount(float* volume, int* count) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int cnt = 0;
	for (int z = 0; z + 1 < VOLUME; z++) {
		cnt += triNumber_device[deviceGetCubeIndex(volume, x, y, z)];
	}
	count[devicePid(x, y)] = cnt;
}

__device__ __forceinline__ float3 deviceCalnEdgePoint(float* volume, int x, int y, int z, int dx, int dy, int dz) {
	float v1 = volume[deviceVid(x, y, z)];
	float v2 = volume[deviceVid(x + dx, y + dy, z + dz)];
	if ((v1 < 0) ^ (v2 < 0)) {
		float k =  v1 / (v1 - v2);
		return make_float3(x + k * dx, y + k * dy, z + k * dz);
	}
	return float3();
}

__device__ __forceinline__ uchar4 calnColor(int cameras, UINT8 bin, float3 ori, Transformation* transformation, Intrinsics* intrinsics, uchar4* colorMap) {
	short4 colorSum = short4();
	int cnt = 0;
	for (int i = 0; i < cameras; i++) {
		if ((bin >> i) & 1) {
			float3 pos = transformation[i].translate(ori);
			int2 pixel = intrinsics[i].translate(pos);
			if (0 <= pixel.x && pixel.x < COLOR_W && 0 <= pixel.y && pixel.y < COLOR_H) {
				cnt++;
				uchar4 tmp = colorMap[(i * COLOR_H + pixel.y) * COLOR_W + pixel.x];
				colorSum.x += tmp.x;
				colorSum.y += tmp.y;
				colorSum.z += tmp.z;
			}
		}
	}
	if (cnt == 0) {
		return uchar4();
	}
	return make_uchar4(colorSum.x / cnt, colorSum.y / cnt, colorSum.z / cnt, 0);
}

__global__ void kernelMarchingCubes(int cameras, float* volume, UINT8* volumeBin, int* count, Vertex* vertex, Transformation* transformation, Intrinsics* intrinsics, uchar4* colorMap, float3 volumeSize, float3 offset) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x + 1 >= VOLUME || y + 1 >= VOLUME) {
		return;
	}	

	float3 pos[12];
	float3 posBuffer[6];
	Vertex* vtx = vertex + count[devicePid(x, y)] * 3;

	for (int z = 0; z + 1 < VOLUME; z++) {
		int cubeId = deviceGetCubeIndex(volume, x, y, z);

		if (triTable_device[cubeId][0] != -1) {
			int id = deviceVid(x, y, z);
			pos[0] = deviceCalnEdgePoint(volume, x + 0, y + 0, z + 0, 1, 0, 0);
			pos[1] = deviceCalnEdgePoint(volume, x + 1, y + 0, z + 0, 0, 1, 0);
			pos[2] = deviceCalnEdgePoint(volume, x + 0, y + 1, z + 0, 1, 0, 0);
			pos[3] = deviceCalnEdgePoint(volume, x + 0, y + 0, z + 0, 0, 1, 0);

			pos[4] = deviceCalnEdgePoint(volume, x + 0, y + 0, z + 1, 1, 0, 0);
			pos[5] = deviceCalnEdgePoint(volume, x + 1, y + 0, z + 1, 0, 1, 0);
			pos[6] = deviceCalnEdgePoint(volume, x + 0, y + 1, z + 1, 1, 0, 0);
			pos[7] = deviceCalnEdgePoint(volume, x + 0, y + 0, z + 1, 0, 1, 0);

			pos[8] = deviceCalnEdgePoint(volume, x + 0, y + 0, z + 0, 0, 0, 1);
			pos[9] = deviceCalnEdgePoint(volume, x + 1, y + 0, z + 0, 0, 0, 1);
			pos[10] = deviceCalnEdgePoint(volume, x + 1, y + 1, z + 0, 0, 0, 1);
			pos[11] = deviceCalnEdgePoint(volume, x + 0, y + 1, z + 0, 0, 0, 1);

			for (int i = 0; i < 5 && triTable_device[cubeId][i * 3] != -1; i++) {
				for (int j = 0; j < 3; j++) {
					int edgeId = triTable_device[cubeId][i * 3 + j];
					posBuffer[j] = pos[edgeId] * volumeSize + offset;
				}
				posBuffer[3] = (posBuffer[0] + posBuffer[1]) * 0.5;
				posBuffer[4] = (posBuffer[1] + posBuffer[2]) * 0.5;
				posBuffer[5] = (posBuffer[2] + posBuffer[0]) * 0.5;
				for (int j = 0; j < 3; j++) {
					vtx->pos = posBuffer[j];
					vtx->color = calnColor(cameras, volumeBin[id], posBuffer[j], transformation, intrinsics, colorMap);
					vtx->color2 = calnColor(cameras, volumeBin[id], posBuffer[j + 3], transformation, intrinsics, colorMap);
					vtx++;
				}
			}
		}
	}
}

__global__ void cudaCountAccumulation(int *count_device, int *sum_device, int *temp_device) {//һ��block��1024���̣߳�����2048������һ����Ҫ����resx*resy = 2^18�������ֳ�128��block��
	int block_offset = blockIdx.x * 2048;//ȷ������ڼ���2048��
	int thid = threadIdx.x;

	__shared__ int shared_count_device[2048];
	if (block_offset == 0 && thid == 0)
		shared_count_device[0] = 0;
	else
		shared_count_device[2 * thid] = count_device[block_offset + 2 * thid - 1];
	shared_count_device[2 * thid + 1] = count_device[block_offset + 2 * thid];
	__syncthreads();//shared��ֵ��Ҫͬ����

	//UpSweep
	int offset = 1;
	int n = 2048;
	for (int d = n >> 1; d > 0; d >>= 1)                    // build sum in place up the tree
	{
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2 * thid + 1) - 1;
			int bi = offset*(2 * thid + 2) - 1;
			shared_count_device[bi] += shared_count_device[ai];
		}
		offset *= 2;
	}

	//DownSweep,ע��������ΪҪ������block����ͣ�����ʹ�õ���inclusive scan��
	for (int i = n / 2; i > 1; i /= 2) {
		__syncthreads();
		int start = (i - 1) + (i >> 1);
		int doffset = (i >> 1);
		if ((2 * thid - start) % i == 0 && 2 * thid - start >= 0) {
			shared_count_device[2 * thid] += shared_count_device[2 * thid - doffset];
		}
		if ((2 * thid + 1 - start) % i == 0 && 2 * thid + 1 - start >= 0) {
			shared_count_device[2 * thid + 1] += shared_count_device[2 * thid + 1 - doffset];
		}
	}
	temp_device[block_offset + 2 * thid] = shared_count_device[2 * thid];
	temp_device[block_offset + 2 * thid + 1] = shared_count_device[2 * thid + 1];
	if (thid == 0) {
		sum_device[blockIdx.x] = shared_count_device[n - 1];
	}
}

__global__ void cudaCountAccumulation2(int *count_device, int *sum_device, int *temp_device) {//һ��block��1024���̣߳�����2048������һ����Ҫ����resx*resy = 2^18�������ֳ�128��block��
	int block_offset = blockIdx.x * 2048;//ȷ������ڼ���2048��
	int thid = threadIdx.x;
	int n = 2048;
	__shared__ int shared_count_device[2048];
	__shared__ int presum;
	shared_count_device[2 * thid] = temp_device[block_offset + 2 * thid];
	shared_count_device[2 * thid + 1] = temp_device[block_offset + 2 * thid + 1];
	if (thid == 0) {
		if (blockIdx.x != 0) {
			presum = sum_device[blockIdx.x - 1];
		}
		else {
			presum = 0;
		}
	}
	__syncthreads();//shared��ֵ��Ҫͬ����

	shared_count_device[2 * thid] += presum;
	shared_count_device[2 * thid + 1] += presum;

	count_device[block_offset + 2 * thid] = shared_count_device[2 * thid];
	count_device[block_offset + 2 * thid + 1] = shared_count_device[2 * thid + 1];
}

int cpu_cudaCountAccumulation() {
	const int DATASIZE = VOLUME * VOLUME;
	int threads = 1024;
	int blocks = DATASIZE / threads / 2;
	int sum_host[128];
	int* sum_device;
	int* temp_device;
	HANDLE_ERROR(hipMalloc(&sum_device, blocks * sizeof(int)));
	HANDLE_ERROR(hipMalloc(&temp_device, DATASIZE * sizeof(int)));
	//stage1
	cudaCountAccumulation << <blocks, threads >> > (count_device, sum_device, temp_device);
	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipMemcpy(sum_host, sum_device, blocks * sizeof(int), hipMemcpyDeviceToHost));
	for (int i = 1; i < blocks; ++i) {
		sum_host[i] += sum_host[i - 1];
	}
	//stage2
	HANDLE_ERROR(hipMemcpy(sum_device, sum_host, blocks * sizeof(int), hipMemcpyHostToDevice));
	cudaCountAccumulation2 << <blocks, threads >> > (count_device, sum_device, temp_device);
	HANDLE_ERROR(hipGetLastError());

	HANDLE_ERROR(hipFree(sum_device));
	HANDLE_ERROR(hipFree(temp_device));
	int tris_size = sum_host[blocks - 1];
	return tris_size;
}

extern "C"
void cudaIntegrate(int cameras, int& triSize, Vertex* vertex, UINT16** depth, float* depth_device, RGBQUAD** color, Transformation* depthTrans, Transformation* colorTrans, Intrinsics* depthIntrinsics, Intrinsics* colorIntrinsics) {
	dim3 blocks = dim3(VOLUME / BLOCK_SIZE, VOLUME / BLOCK_SIZE);
	dim3 threads = dim3(BLOCK_SIZE, BLOCK_SIZE);

	HANDLE_ERROR(hipMemcpy(depthTrans_device, depthTrans, MAX_CAMERAS * sizeof(Transformation), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(depthIntrinsics_device, depthIntrinsics, MAX_CAMERAS * sizeof(Intrinsics), hipMemcpyHostToDevice));
	/*for (int i = 0; i < cameras; i++) {
		if (depth[i] != NULL) {
			HANDLE_ERROR(hipMemcpy(depth_device + i * DEPTH_W * DEPTH_H, depth[i], DEPTH_W * DEPTH_H * sizeof(UINT16), hipMemcpyHostToDevice));
		}
	}*/

	HANDLE_ERROR(hipMemcpyAsync(colorTrans_device, colorTrans, MAX_CAMERAS * sizeof(Transformation), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyAsync(colorIntrinsics_device, colorIntrinsics, MAX_CAMERAS * sizeof(Intrinsics), hipMemcpyHostToDevice));
	for (int i = 0; i < cameras; i++) {
		if (color[i] != NULL) {
			HANDLE_ERROR(hipMemcpyAsync(color_device + i * COLOR_W * COLOR_H, color[i], COLOR_W * COLOR_H * sizeof(uchar4), hipMemcpyHostToDevice));
		}
	}

	kernelIntegrateDepth << <blocks, threads >> > (cameras, volume_device, volumeBin_device, depthTrans_device, depthIntrinsics_device, depth_device, volumeSize, offset); //1.7ms
	HANDLE_ERROR(hipGetLastError());

	kernelMarchingCubesCount << <blocks, threads >> > (volume_device, count_device);
	HANDLE_ERROR(hipGetLastError());
	triSize = cpu_cudaCountAccumulation();

	if (triSize * 3 <= MAX_VERTEX) {
		kernelMarchingCubes << <blocks, threads >> > (cameras, volume_device, volumeBin_device, count_device, vertex_device, colorTrans_device, colorIntrinsics_device, color_device, volumeSize, offset); //3.6ms
		HANDLE_ERROR(hipGetLastError());
		HANDLE_ERROR(hipMemcpy(vertex, vertex_device, triSize * 3 * sizeof(Vertex), hipMemcpyDeviceToHost));
	} else {
		std::cout << "vertex size limit exceeded" << std::endl;
	}
}
