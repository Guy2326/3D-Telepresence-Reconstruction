#include "hip/hip_runtime.h"
#include "CudaHandleError.h"
#include "Parameters.h"
#include <Windows.h>
#include <iostream>
#include "Timer.h"
#include "Vertex.h"

#define BLOCK_SIZE 16 // FIXED !!! for optimization
#define BLOCK_LOG 4
#define MAX_CAMERAS 8

namespace tsdf {
	int3 resolution;
	float3 size;
	float3 center;
	float3 volumeSize;
	float3 offset;

	float* volume_device;
	uchar4* volume_color_device;
	UINT8* cubeIndex_device;
	hipChannelFormatDesc depthDesc;
	hipChannelFormatDesc colorDesc;
	hipArray* depth_device;
	hipArray* color_device;
	texture<UINT16, 2, hipReadModeElementType> depthTexture;
	texture<uchar4, 2, hipReadModeElementType> colorTexture;
	float* transformation_device;
	int* count_device;
	int* count_host;

	dim3 grid;
	dim3 block;
}
using namespace tsdf;

__device__ __forceinline__ int devicePid(int x, int y, int3 resolution) {
	int bx = x >> BLOCK_LOG, tx = x ^ (bx << BLOCK_LOG);
	int by = y >> BLOCK_LOG, ty = y ^ (by << BLOCK_LOG);
	return ((((by * gridDim.x + bx) << BLOCK_LOG) + ty) << BLOCK_LOG) + tx;
}

__device__ __forceinline__ int deviceVid(int x, int y, int z, int3 resolution) {
	return devicePid(x, y, resolution) + z * resolution.x * resolution.y;
}

extern "C"
void cudaInitVolume(int resolutionX, int resolutionY, int resolutionZ, float sizeX, float sizeY, float sizeZ, float centerX, float centerY, float centerZ) {
	resolution.x = resolutionX;
	resolution.y = resolutionY;
	resolution.z = resolutionZ;
	size.x = sizeX;
	size.y = sizeY;
	size.z = sizeZ;
	center.x = centerX;
	center.y = centerY;
	center.z = centerZ;
	volumeSize.x = size.x / resolution.x;
	volumeSize.y = size.y / resolution.y;
	volumeSize.z = size.z / resolution.z;
	offset.x = center.x - size.x / 2;
	offset.y = center.y - size.y / 2;
	offset.z = center.z - size.z / 2;
	HANDLE_ERROR(hipMalloc(&volume_device, resolution.x * resolution.y * resolution.z * sizeof(float)));
	HANDLE_ERROR(hipMalloc(&volume_color_device, resolution.x * resolution.y * resolution.z * sizeof(uchar4)));
	HANDLE_ERROR(hipMalloc(&cubeIndex_device, resolution.x * resolution.y * resolution.z * sizeof(UINT8)));
	depthDesc = hipCreateChannelDesc<UINT16>();
	HANDLE_ERROR(hipMallocArray(&depth_device, &depthDesc, DEPTH_W, DEPTH_H));
	colorDesc = hipCreateChannelDesc<uchar4>();
	HANDLE_ERROR(hipMallocArray(&color_device, &colorDesc, COLOR_W, COLOR_H));

	HANDLE_ERROR(hipMalloc(&transformation_device, 16 * sizeof(float) * MAX_CAMERAS));
	HANDLE_ERROR(hipMalloc(&count_device, resolution.x * resolution.y * sizeof(int)));
	count_host = new int[resolution.x * resolution.y];
	block = dim3(BLOCK_SIZE, BLOCK_SIZE);
	grid = dim3((resolution.x + BLOCK_SIZE - 1) / BLOCK_SIZE, (resolution.y + BLOCK_SIZE - 1) / BLOCK_SIZE);
}

extern "C"
void cudaReleaseVolume() {
	HANDLE_ERROR(hipFree(volume_device));
	HANDLE_ERROR(hipFree(volume_color_device));
	HANDLE_ERROR(hipFree(cubeIndex_device));
	HANDLE_ERROR(hipFreeArray(depth_device));
	HANDLE_ERROR(hipFreeArray(color_device));
	HANDLE_ERROR(hipFree(transformation_device));
	HANDLE_ERROR(hipFree(count_device));
	delete[] count_host;
}

__global__ void kernelIntegrateDepth(int cameras, float* volume, uchar4* volume_color, float* transformation, int3 resolution, float3 volumeSize, float3 offset) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= resolution.x || y >= resolution.y) {
		return;
	}

	const float TRANC_DIST_M = 3.1 * max(volumeSize.x, max(volumeSize.y, volumeSize.z));
	uchar4 color;
	color.x = color.y = color.z = 255;

	float oriX = (x + 0.5) * volumeSize.x + offset.x;
	float oriY = (y + 0.5) * volumeSize.y + offset.y;
	float oriZ = (0 - 0.5) * volumeSize.z + offset.z;
	float posX = transformation[0] * oriX + transformation[1] * oriY + transformation[2] * oriZ + transformation[12];
	float posY = transformation[4] * oriX + transformation[5] * oriY + transformation[6] * oriZ + transformation[13];
	float posZ = transformation[8] * oriX + transformation[9] * oriY + transformation[10] * oriZ + transformation[14];
	const float deltaX = transformation[2] * volumeSize.z;
	const float deltaY = transformation[6] * volumeSize.z;
	const float deltaZ = transformation[10] * volumeSize.z;

	__syncthreads();

	for (int z = 0; z < resolution.z; z++) {
		bool flag = false;
		float tsdf = 1.0;

		posX += deltaX;
		posY += deltaY;
		posZ += deltaZ;

		float depthX = posX * DEPTH_FX / posZ + DEPTH_CX;
		float depthY = posY * DEPTH_FY / posZ + DEPTH_CY;
			
		if (posZ > 0 && 0 <= depthX && depthX <= DEPTH_W && 0 <= depthY && depthY <= DEPTH_H) {
			UINT16 depth = tex2D<UINT16>(depthTexture, depthX, depthY);

			if (depth != 0) {
				float sdf = depth * 0.001 - posZ;

				if (sdf >= -TRANC_DIST_M) {
					flag = true;
					sdf = sdf / TRANC_DIST_M;
					if (sdf < tsdf) {
						float colorX = posX * COLOR_FX / posZ + COLOR_CX;
						float colorY = posY * COLOR_FY / posZ + COLOR_CY;
						if ( 0 <= colorX && colorX <= COLOR_W && 0 <= colorY && colorY <= COLOR_H) {
							color = tex2D<uchar4>(colorTexture, colorX, colorY);
						}
						tsdf = sdf;
					}
				}
			}
		}

		int id = deviceVid(x, y, z, resolution);
		__syncthreads();

		if (flag) {
			volume[id] = tsdf;
		} else {
			volume[id] = -1;
		}
		__syncthreads();

		if (flag) {
			volume_color[id] = color;
		}
		__syncthreads();
	}
}

extern "C"
void cudaIntegrateDepth(int cameras, UINT16** depth, RGBQUAD** color, float** transformation) {
	HANDLE_ERROR(hipMemcpy(transformation_device + 0 * 16, transformation[0], 16 * sizeof(float), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpyToArray(depth_device, 0, 0, depth[0], sizeof(UINT16) * DEPTH_W * DEPTH_H, hipMemcpyHostToDevice));
	depthTexture.filterMode = hipFilterModePoint;
	depthTexture.addressMode[0] = hipAddressModeWrap;
	depthTexture.addressMode[1] = hipAddressModeWrap;
	HANDLE_ERROR(hipBindTextureToArray(&depthTexture, depth_device, &depthDesc));

	HANDLE_ERROR(hipMemcpyToArray(color_device, 0, 0, color[0], sizeof(uchar4) * COLOR_W * COLOR_H, hipMemcpyHostToDevice));
	colorTexture.filterMode = hipFilterModePoint;
	colorTexture.addressMode[0] = hipAddressModeWrap;
	colorTexture.addressMode[1] = hipAddressModeWrap;
	HANDLE_ERROR(hipBindTextureToArray(&colorTexture, color_device, &colorDesc));

	kernelIntegrateDepth << <grid, block >> > (cameras, volume_device, volume_color_device, transformation_device, resolution, volumeSize, offset);
	HANDLE_ERROR(hipGetLastError());

	HANDLE_ERROR(hipUnbindTexture(&depthTexture));
	HANDLE_ERROR(hipUnbindTexture(&colorTexture));
}

__constant__ UINT8 triNumber_device[256] = {0, 1, 1, 2, 1, 2, 2, 3, 1, 2, 2, 3, 2, 3, 3, 2, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 2, 3, 4, 4, 3, 3, 4, 4, 3, 4, 5, 5, 2, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 4, 2, 3, 3, 4, 3, 4, 2, 3, 3, 4, 4, 5, 4, 5, 3, 2, 3, 4, 4, 3, 4, 5, 3, 2, 4, 5, 5, 4, 5, 2, 4, 1, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 4, 3, 4, 4, 5, 3, 2, 4, 3, 4, 3, 5, 2, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 4, 3, 4, 4, 3, 4, 5, 5, 4, 4, 3, 5, 2, 5, 4, 2, 1, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 2, 3, 3, 2, 3, 4, 4, 5, 4, 5, 5, 2, 4, 3, 5, 4, 3, 2, 4, 1, 3, 4, 4, 5, 4, 5, 3, 4, 4, 5, 5, 2, 3, 4, 2, 1, 2, 3, 3, 2, 3, 4, 2, 1, 3, 2, 4, 1, 2, 1, 1, 0};
__constant__ INT8 triTable_device[256][16] =
{ { -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 3, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 2, 10, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 8, 3, 2, 10, 8, 10, 9, 8, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 11, 2, 8, 11, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 9, 0, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 11, 2, 1, 9, 11, 9, 8, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 10, 1, 11, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 10, 1, 0, 8, 10, 8, 11, 10, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 9, 0, 3, 11, 9, 11, 10, 9, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 10, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 4, 7, 3, 0, 4, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 2, 10, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 10, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1, -1 },
{ 8, 4, 7, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 11, 4, 7, 11, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 0, 1, 8, 4, 7, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 7, 11, 9, 4, 11, 9, 11, 2, 9, 2, 1, -1, -1, -1, -1 },
{ 3, 10, 1, 3, 11, 10, 7, 8, 4, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 11, 10, 1, 4, 11, 1, 0, 4, 7, 11, 4, -1, -1, -1, -1 },
{ 4, 7, 8, 9, 0, 11, 9, 11, 10, 11, 0, 3, -1, -1, -1, -1 },
{ 4, 7, 11, 4, 11, 9, 9, 11, 10, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 10, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 0, 8, 1, 2, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 2, 10, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 10, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1, -1 },
{ 9, 5, 4, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 11, 2, 0, 8, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 5, 4, 0, 1, 5, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 1, 5, 2, 5, 8, 2, 8, 11, 4, 8, 5, -1, -1, -1, -1 },
{ 10, 3, 11, 10, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 9, 5, 0, 8, 1, 8, 10, 1, 8, 11, 10, -1, -1, -1, -1 },
{ 5, 4, 0, 5, 0, 11, 5, 11, 10, 11, 0, 3, -1, -1, -1, -1 },
{ 5, 4, 8, 5, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 7, 8, 9, 5, 7, 10, 1, 2, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1, -1 },
{ 8, 0, 2, 8, 2, 5, 8, 5, 7, 10, 5, 2, -1, -1, -1, -1 },
{ 2, 10, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 7, 9, 5, 7, 8, 9, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 11, -1, -1, -1, -1 },
{ 2, 3, 11, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1, -1 },
{ 11, 2, 1, 11, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 5, 8, 8, 5, 7, 10, 1, 3, 10, 3, 11, -1, -1, -1, -1 },
{ 5, 7, 0, 5, 0, 9, 7, 11, 0, 1, 0, 10, 11, 10, 0, -1 },
{ 11, 10, 0, 11, 0, 3, 10, 5, 0, 8, 0, 7, 5, 7, 0, -1 },
{ 11, 10, 5, 7, 11, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 3, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 0, 1, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 8, 3, 1, 9, 8, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1, -1 },
{ 2, 3, 11, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 11, 0, 8, 11, 2, 0, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 1, 9, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 10, 6, 1, 9, 2, 9, 11, 2, 9, 8, 11, -1, -1, -1, -1 },
{ 6, 3, 11, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 11, 0, 11, 5, 0, 5, 1, 5, 11, 6, -1, -1, -1, -1 },
{ 3, 11, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1, -1 },
{ 6, 5, 9, 6, 9, 11, 11, 9, 8, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 10, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 3, 0, 4, 7, 3, 6, 5, 10, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 9, 0, 5, 10, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1, -1 },
{ 6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1, -1 },
{ 8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1, -1 },
{ 7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9, -1 },
{ 3, 11, 2, 7, 8, 4, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 10, 6, 4, 7, 2, 4, 2, 0, 2, 7, 11, -1, -1, -1, -1 },
{ 0, 1, 9, 4, 7, 8, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1 },
{ 9, 2, 1, 9, 11, 2, 9, 4, 11, 7, 11, 4, 5, 10, 6, -1 },
{ 8, 4, 7, 3, 11, 5, 3, 5, 1, 5, 11, 6, -1, -1, -1, -1 },
{ 5, 1, 11, 5, 11, 6, 1, 0, 11, 7, 11, 4, 0, 4, 11, -1 },
{ 0, 5, 9, 0, 6, 5, 0, 3, 6, 11, 6, 3, 8, 4, 7, -1 },
{ 6, 5, 9, 6, 9, 11, 4, 7, 9, 7, 11, 9, -1, -1, -1, -1 },
{ 10, 4, 9, 6, 4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 10, 6, 4, 9, 10, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 0, 1, 10, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 10, -1, -1, -1, -1 },
{ 1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1, -1 },
{ 0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 4, 9, 10, 6, 4, 11, 2, 3, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 2, 2, 8, 11, 4, 9, 10, 4, 10, 6, -1, -1, -1, -1 },
{ 3, 11, 2, 0, 1, 6, 0, 6, 4, 6, 1, 10, -1, -1, -1, -1 },
{ 6, 4, 1, 6, 1, 10, 4, 8, 1, 2, 1, 11, 8, 11, 1, -1 },
{ 9, 6, 4, 9, 3, 6, 9, 1, 3, 11, 6, 3, -1, -1, -1, -1 },
{ 8, 11, 1, 8, 1, 0, 11, 6, 1, 9, 1, 4, 6, 4, 1, -1 },
{ 3, 11, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1, -1 },
{ 6, 4, 8, 11, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 7, 10, 6, 7, 8, 10, 8, 9, 10, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 7, 3, 0, 10, 7, 0, 9, 10, 6, 7, 10, -1, -1, -1, -1 },
{ 10, 6, 7, 1, 10, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1, -1 },
{ 10, 6, 7, 10, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1, -1 },
{ 2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9, -1 },
{ 7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1, -1 },
{ 7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 3, 11, 10, 6, 8, 10, 8, 9, 8, 6, 7, -1, -1, -1, -1 },
{ 2, 0, 7, 2, 7, 11, 0, 9, 7, 6, 7, 10, 9, 10, 7, -1 },
{ 1, 8, 0, 1, 7, 8, 1, 10, 7, 6, 7, 10, 2, 3, 11, -1 },
{ 11, 2, 1, 11, 1, 7, 10, 6, 1, 6, 7, 1, -1, -1, -1, -1 },
{ 8, 9, 6, 8, 6, 7, 9, 1, 6, 11, 6, 3, 1, 3, 6, -1 },
{ 0, 9, 1, 11, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 7, 8, 0, 7, 0, 6, 3, 11, 0, 11, 6, 0, -1, -1, -1, -1 },
{ 7, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 0, 8, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 1, 9, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 1, 9, 8, 3, 1, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 1, 2, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 10, 3, 0, 8, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 9, 0, 2, 10, 9, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 6, 11, 7, 2, 10, 3, 10, 8, 3, 10, 9, 8, -1, -1, -1, -1 },
{ 7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1, -1 },
{ 10, 7, 6, 10, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 7, 6, 1, 7, 10, 1, 8, 7, 1, 0, 8, -1, -1, -1, -1 },
{ 0, 3, 7, 0, 7, 10, 0, 10, 9, 6, 10, 7, -1, -1, -1, -1 },
{ 7, 6, 10, 7, 10, 8, 8, 10, 9, -1, -1, -1, -1, -1, -1, -1 },
{ 6, 8, 4, 11, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 6, 11, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 6, 11, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 4, 6, 9, 6, 3, 9, 3, 1, 11, 3, 6, -1, -1, -1, -1 },
{ 6, 8, 4, 6, 11, 8, 2, 10, 1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 10, 3, 0, 11, 0, 6, 11, 0, 4, 6, -1, -1, -1, -1 },
{ 4, 11, 8, 4, 6, 11, 0, 2, 9, 2, 10, 9, -1, -1, -1, -1 },
{ 10, 9, 3, 10, 3, 2, 9, 4, 3, 11, 3, 6, 4, 6, 3, -1 },
{ 8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1, -1 },
{ 1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 10, 1, -1, -1, -1, -1 },
{ 10, 1, 0, 10, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 6, 3, 4, 3, 8, 6, 10, 3, 0, 3, 9, 10, 9, 3, -1 },
{ 10, 9, 4, 6, 10, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 9, 5, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 3, 4, 9, 5, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 0, 1, 5, 4, 0, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 11, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1, -1 },
{ 9, 5, 4, 10, 1, 2, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 6, 11, 7, 1, 2, 10, 0, 8, 3, 4, 9, 5, -1, -1, -1, -1 },
{ 7, 6, 11, 5, 4, 10, 4, 2, 10, 4, 0, 2, -1, -1, -1, -1 },
{ 3, 4, 8, 3, 5, 4, 3, 2, 5, 10, 5, 2, 11, 7, 6, -1 },
{ 7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1, -1 },
{ 3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1, -1 },
{ 6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8, -1 },
{ 9, 5, 4, 10, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1, -1 },
{ 1, 6, 10, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4, -1 },
{ 4, 0, 10, 4, 10, 5, 0, 3, 10, 6, 10, 7, 3, 7, 10, -1 },
{ 7, 6, 10, 7, 10, 8, 5, 4, 10, 4, 8, 10, -1, -1, -1, -1 },
{ 6, 9, 5, 6, 11, 9, 11, 8, 9, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 6, 11, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1, -1 },
{ 0, 11, 8, 0, 5, 11, 0, 1, 5, 5, 6, 11, -1, -1, -1, -1 },
{ 6, 11, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 10, 9, 5, 11, 9, 11, 8, 11, 5, 6, -1, -1, -1, -1 },
{ 0, 11, 3, 0, 6, 11, 0, 9, 6, 5, 6, 9, 1, 2, 10, -1 },
{ 11, 8, 5, 11, 5, 6, 8, 0, 5, 10, 5, 2, 0, 2, 5, -1 },
{ 6, 11, 3, 6, 3, 5, 2, 10, 3, 10, 5, 3, -1, -1, -1, -1 },
{ 5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1, -1 },
{ 9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8, -1 },
{ 1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 3, 6, 1, 6, 10, 3, 8, 6, 5, 6, 9, 8, 9, 6, -1 },
{ 10, 1, 0, 10, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1, -1 },
{ 0, 3, 8, 5, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 11, 5, 10, 7, 5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 11, 5, 10, 11, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 11, 7, 5, 10, 11, 1, 9, 0, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 7, 5, 10, 11, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1, -1 },
{ 11, 1, 2, 11, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 11, -1, -1, -1, -1 },
{ 9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 11, 7, -1, -1, -1, -1 },
{ 7, 5, 2, 7, 2, 11, 5, 9, 2, 3, 2, 8, 9, 8, 2, -1 },
{ 2, 5, 10, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 2, 0, 8, 5, 2, 8, 7, 5, 10, 2, 5, -1, -1, -1, -1 },
{ 9, 0, 1, 5, 10, 3, 5, 3, 7, 3, 10, 2, -1, -1, -1, -1 },
{ 9, 8, 2, 9, 2, 1, 8, 7, 2, 10, 2, 5, 7, 5, 2, -1 },
{ 1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 8, 4, 5, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 0, 4, 5, 11, 0, 5, 10, 11, 11, 3, 0, -1, -1, -1, -1 },
{ 0, 1, 9, 8, 4, 10, 8, 10, 11, 10, 4, 5, -1, -1, -1, -1 },
{ 10, 11, 4, 10, 4, 5, 11, 3, 4, 9, 4, 1, 3, 1, 4, -1 },
{ 2, 5, 1, 2, 8, 5, 2, 11, 8, 4, 5, 8, -1, -1, -1, -1 },
{ 0, 4, 11, 0, 11, 3, 4, 5, 11, 2, 11, 1, 5, 1, 11, -1 },
{ 0, 2, 5, 0, 5, 9, 2, 11, 5, 4, 5, 8, 11, 8, 5, -1 },
{ 9, 4, 5, 2, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 5, 10, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1, -1 },
{ 5, 10, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 10, 2, 3, 5, 10, 3, 8, 5, 4, 5, 8, 0, 1, 9, -1 },
{ 5, 10, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1, -1 },
{ 8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1, -1 },
{ 9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 11, 7, 4, 9, 11, 9, 10, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 3, 4, 9, 7, 9, 11, 7, 9, 10, 11, -1, -1, -1, -1 },
{ 1, 10, 11, 1, 11, 4, 1, 4, 0, 7, 4, 11, -1, -1, -1, -1 },
{ 3, 1, 4, 3, 4, 8, 1, 10, 4, 7, 4, 11, 10, 11, 4, -1 },
{ 4, 11, 7, 9, 11, 4, 9, 2, 11, 9, 1, 2, -1, -1, -1, -1 },
{ 9, 7, 4, 9, 11, 7, 9, 1, 11, 2, 11, 1, 0, 8, 3, -1 },
{ 11, 7, 4, 11, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1, -1 },
{ 11, 7, 4, 11, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1, -1 },
{ 2, 9, 10, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1, -1 },
{ 9, 10, 7, 9, 7, 4, 10, 2, 7, 8, 7, 0, 2, 0, 7, -1 },
{ 3, 7, 10, 3, 10, 2, 7, 4, 10, 1, 10, 0, 4, 0, 10, -1 },
{ 1, 10, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1, -1 },
{ 4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 0, 9, 3, 9, 11, 11, 9, 10, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 1, 10, 0, 10, 8, 8, 10, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 1, 10, 11, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 11, 1, 11, 9, 9, 11, 8, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 0, 9, 3, 9, 11, 1, 2, 9, 2, 11, 9, -1, -1, -1, -1 },
{ 0, 2, 11, 8, 0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 3, 8, 2, 8, 10, 10, 8, 9, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 10, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 3, 8, 2, 8, 10, 0, 1, 8, 1, 10, 8, -1, -1, -1, -1 },
{ 1, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 } };

__device__ __forceinline__ UINT16 deviceGetCubeIndex(float* volume, int x, int y, int z, int3 resolution) {
	if (x + 1 >= resolution.x) return 0;
	if (y + 1 >= resolution.y) return 0;
	if (z + 1 >= resolution.z) return 0;
	if (volume[deviceVid(x + 0, y + 0, z + 0, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 1, y + 0, z + 0, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 0, y + 1, z + 0, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 1, y + 1, z + 0, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 0, y + 0, z + 1, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 1, y + 0, z + 1, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 0, y + 1, z + 1, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 1, y + 1, z + 1, resolution)] == -1) return 0;
	UINT16 index = 0;
	if (volume[deviceVid(x + 0, y + 0, z + 0, resolution)] < 0) index |= 1;
	if (volume[deviceVid(x + 1, y + 0, z + 0, resolution)] < 0) index |= 2;
	if (volume[deviceVid(x + 0, y + 1, z + 0, resolution)] < 0) index |= 8;
	if (volume[deviceVid(x + 1, y + 1, z + 0, resolution)] < 0) index |= 4;
	if (volume[deviceVid(x + 0, y + 0, z + 1, resolution)] < 0) index |= 16;
	if (volume[deviceVid(x + 1, y + 0, z + 1, resolution)] < 0) index |= 32;
	if (volume[deviceVid(x + 0, y + 1, z + 1, resolution)] < 0) index |= 128;
	if (volume[deviceVid(x + 1, y + 1, z + 1, resolution)] < 0) index |= 64;
	return index;
}

__global__ void kernelMarchingCubesCount(float* volume, UINT8* cubeIndex, int* count, int3 resolution) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int cnt = 0;
	for (int z = 0; z + 1 < resolution.z; z++) {
		int id = deviceVid(x, y, z, resolution);
		int cubeId = cubeIndex[id] = deviceGetCubeIndex(volume, x, y, z, resolution);
		cnt += triNumber_device[cubeId];
		__syncthreads();
	}

	__syncthreads();
	count[devicePid(x, y, resolution)] = cnt;
}

__device__ __forceinline__ void deviceCalnEdgePoint(float* volume, uchar4* volume_color, int x1, int y1, int z1, int x2, int y2, int z2, float3& pos, uchar4& color, int3 resolution, float3 volumeSize, float3 offset) {
	int id1 = deviceVid(x1, y1, z1, resolution);
	int id2 = deviceVid(x2, y2, z2, resolution);
	float v1 = volume[id1];
	float v2 = volume[id2];
	if ((v1 < 0) ^ (v2 < 0)) {
		float k =  v1 / (v1 - v2);
		pos.x = ((1 - k) * x1 + k * x2 - 0.5) * volumeSize.x + offset.x;
		pos.y = ((1 - k) * y1 + k * y2 - 0.5) * volumeSize.y + offset.y;
		pos.z = ((1 - k) * z1 + k * z2 - 0.5) * volumeSize.z + offset.z;
		color.x = (UINT8)min((1 - k) * volume_color[id1].x + k * volume_color[id2].x, 255.0);
		color.y = (UINT8)min((1 - k) * volume_color[id1].y + k * volume_color[id2].y, 255.0);
		color.z = (UINT8)min((1 - k) * volume_color[id1].z + k * volume_color[id2].z, 255.0);
	}
}

__global__ void kernelMarchingCubes(float* volume, uchar4* volume_color, UINT8* cubeIndex, int* count, Vertex* vertex, int3 resolution, float3 volumeSize, float3 offset) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x + 1 >= resolution.x || y + 1 >= resolution.y) {
		return;
	}	

	float3 pos[12];
	uchar4 color[12];

	const int MAX_BUFFER = 512;
	int tot = 0;
	float3 posBuffer[MAX_BUFFER];
	uchar4 colorBuffer[MAX_BUFFER];

	for (int z = 0; z + 1 < resolution.z; z++) {
		int id = deviceVid(x, y, z, resolution);
		int cubeId = cubeIndex[id];

		deviceCalnEdgePoint(volume, volume_color, x + 0, y + 0, z + 0, x + 1, y + 0, z + 0, pos[0], color[0], resolution, volumeSize, offset);
		deviceCalnEdgePoint(volume, volume_color, x + 1, y + 0, z + 0, x + 1, y + 1, z + 0, pos[1], color[1], resolution, volumeSize, offset);
		deviceCalnEdgePoint(volume, volume_color, x + 1, y + 1, z + 0, x + 0, y + 1, z + 0, pos[2], color[2], resolution, volumeSize, offset);
		deviceCalnEdgePoint(volume, volume_color, x + 0, y + 1, z + 0, x + 0, y + 0, z + 0, pos[3], color[3], resolution, volumeSize, offset);
		
		deviceCalnEdgePoint(volume, volume_color, x + 0, y + 0, z + 1, x + 1, y + 0, z + 1, pos[4], color[4], resolution, volumeSize, offset);
		deviceCalnEdgePoint(volume, volume_color, x + 1, y + 0, z + 1, x + 1, y + 1, z + 1, pos[5], color[5], resolution, volumeSize, offset);
		deviceCalnEdgePoint(volume, volume_color, x + 1, y + 1, z + 1, x + 0, y + 1, z + 1, pos[6], color[6], resolution, volumeSize, offset);
		deviceCalnEdgePoint(volume, volume_color, x + 0, y + 1, z + 1, x + 0, y + 0, z + 1, pos[7], color[7], resolution, volumeSize, offset);

		deviceCalnEdgePoint(volume, volume_color, x + 0, y + 0, z + 0, x + 0, y + 0, z + 1, pos[8], color[8], resolution, volumeSize, offset);
		deviceCalnEdgePoint(volume, volume_color, x + 1, y + 0, z + 0, x + 1, y + 0, z + 1, pos[9], color[9], resolution, volumeSize, offset);
		deviceCalnEdgePoint(volume, volume_color, x + 1, y + 1, z + 0, x + 1, y + 1, z + 1, pos[10], color[10], resolution, volumeSize, offset);
		deviceCalnEdgePoint(volume, volume_color, x + 0, y + 1, z + 0, x + 0, y + 1, z + 1, pos[11], color[11], resolution, volumeSize, offset);
		
		if (triTable_device[cubeId][0] != -1) {
			for (int i = 0; i < 5; i++) {
				if (triTable_device[cubeId][i * 3] != -1) {
					for (int j = 0; j < 3; j++) {
						int edgeId = triTable_device[cubeId][i * 3 + j];
						posBuffer[tot] = pos[edgeId];
						colorBuffer[tot] = color[edgeId];
						tot++;
					}
				} else {
					break;
				}
			}
		}

		__syncthreads();
	}

	int tid = count[devicePid(x, y, resolution)] * 3;
	for (int i = 0; i < tot; i++) {
		vertex[tid + i].pos = posBuffer[i];
		vertex[tid + i].color = colorBuffer[i];
	}
}

__global__ void cudaCountAccumulation(int *count_device, int *sum_device, int *temp_device) {//һ��block��1024���̣߳�����2048������һ����Ҫ����resx*resy = 2^18�������ֳ�128��block��fixed����
	int block_offset = blockIdx.x * 2048;//ȷ������ڼ���2048��
	int thid = threadIdx.x;

	__shared__ int shared_count_device[2048];
	if (block_offset == 0 && thid == 0)
		shared_count_device[0] = 0;
	else
		shared_count_device[2 * thid] = count_device[block_offset + 2 * thid - 1];
	shared_count_device[2 * thid + 1] = count_device[block_offset + 2 * thid];
	__syncthreads();//shared��ֵ��Ҫͬ����

					//UpSweep
	int offset = 1;
	int n = 2048;
	for (int d = n >> 1; d > 0; d >>= 1)                    // build sum in place up the tree
	{
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2 * thid + 1) - 1;
			int bi = offset*(2 * thid + 2) - 1;
			shared_count_device[bi] += shared_count_device[ai];
		}
		offset *= 2;
	}

	//DownSweep,ע��������ΪҪ������block����ͣ�����ʹ�õ���inclusive scan�����ǽ̳��е�exclusive scan��
	for (int i = n / 2; i > 1; i /= 2) {
		__syncthreads();
		int start = (i - 1) + (i >> 1);
		int doffset = (i >> 1);
		if ((2 * thid - start) % i == 0 && 2 * thid - start >= 0) {
			shared_count_device[2 * thid] += shared_count_device[2 * thid - doffset];
		}
		if ((2 * thid + 1 - start) % i == 0 && 2 * thid + 1 - start >= 0) {
			shared_count_device[2 * thid + 1] += shared_count_device[2 * thid + 1 - doffset];
		}
	}
	temp_device[block_offset + 2 * thid] = shared_count_device[2 * thid];
	temp_device[block_offset + 2 * thid + 1] = shared_count_device[2 * thid + 1];
	if (thid == 0) {
		sum_device[blockIdx.x] = shared_count_device[n - 1];
	}
}

__global__ void cudaCountAccumulation2(int *count_device, int *sum_device, int *temp_device) {//һ��block��1024���̣߳�����2048������һ����Ҫ����resx*resy = 2^18�������ֳ�128��block��fixed����
	int block_offset = blockIdx.x * 2048;//ȷ������ڼ���2048��
	int thid = threadIdx.x;
	int n = 2048;
	__shared__ int shared_count_device[2048];
	__shared__ int presum;
	shared_count_device[2 * thid] = temp_device[block_offset + 2 * thid];
	shared_count_device[2 * thid + 1] = temp_device[block_offset + 2 * thid + 1];
	if (blockIdx.x == 0 && thid == 0) {
		count_device[0] = count_device[114688];
	}
	if (thid == 0) {
		if (blockIdx.x != 0) {
			presum = sum_device[blockIdx.x - 1];
		}
		else {
			presum = 0;
		}
	}
	__syncthreads();//shared��ֵ��Ҫͬ����

	shared_count_device[2 * thid] += presum;
	shared_count_device[2 * thid + 1] += presum;

	count_device[block_offset + 2 * thid] = shared_count_device[2 * thid];
	count_device[block_offset + 2 * thid + 1] = shared_count_device[2 * thid + 1];
}

int cpu_cudaCountAccumulation() {
	int temp_grid = 128, temp_block = 1024;
	const int DATASIZE = 262144;
	int *temp_device;
	int *sum_device, *sum_host;
	HANDLE_ERROR(hipMalloc(&temp_device, DATASIZE * sizeof(int)));
	HANDLE_ERROR(hipMalloc(&sum_device, temp_grid * sizeof(int)));
	sum_host = new int[temp_grid];
	for (int i = 0; i < temp_grid; ++i) {
		sum_host[i] = 0;
	}
	//stage1
	HANDLE_ERROR(hipMemcpy(sum_device, sum_host, temp_grid * sizeof(int), hipMemcpyHostToDevice));
	cudaCountAccumulation << <temp_grid, temp_block >> > (count_device, sum_device, temp_device);
	HANDLE_ERROR(hipMemcpy(sum_host, sum_device, temp_grid * sizeof(int), hipMemcpyDeviceToHost));
	for (int i = 1; i<temp_grid; ++i) {
		sum_host[i] += sum_host[(i - 1)];
	}
	//stage2
	HANDLE_ERROR(hipMemcpy(sum_device, sum_host, temp_grid * sizeof(int), hipMemcpyHostToDevice));
	cudaCountAccumulation2 << <temp_grid, temp_block >> > (count_device, sum_device, temp_device);

	int tris_size = sum_host[temp_grid - 1];
	return tris_size;
}

extern "C"
void cudaCalculateMesh(Vertex* vertex, int& tri_size) {
	kernelMarchingCubesCount << <grid, block >> > (volume_device, cubeIndex_device, count_device, resolution);

	hipDeviceSynchronize();
	Timer timer;

	HANDLE_ERROR(hipGetLastError());
	tri_size = cpu_cudaCountAccumulation();
	HANDLE_ERROR(hipGetLastError());

	hipDeviceSynchronize();
	timer.outputTime();

	Vertex* vertex_device;
	HANDLE_ERROR(hipMalloc(&vertex_device, tri_size * 3 * sizeof(Vertex)));
	kernelMarchingCubes << <grid, block >> > (volume_device, volume_color_device, cubeIndex_device, count_device, vertex_device, resolution, volumeSize, offset);

	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipMemcpy(vertex, vertex_device, tri_size * 3 * sizeof(Vertex), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipFree(vertex_device));
}
