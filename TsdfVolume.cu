#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <Windows.h>
#include <iostream>
#include "Timer.h"

#define BLOCK_SIZE 16

namespace tsdf {
	const int W = 512;
	const int H = 424;
	
	int3 resolution;
	float3 size;
	float3 center;

	float* volume_device;
	UINT8* volume_color_device;
	UINT16* depth_device;
	UINT8* color_device;
	float* transformation_device;
	int* count_device;
	int* count_host;

	dim3 grid;
	dim3 block;
}
using namespace tsdf;

__device__ __forceinline__ int devicePid(int x, int y, int3 resolution) {
	int gx = gridDim.x, bx = x / BLOCK_SIZE, tx = x % BLOCK_SIZE;
	int gy = gridDim.y, by = y / BLOCK_SIZE, ty = y % BLOCK_SIZE;
	return (by * gx + bx) * BLOCK_SIZE * BLOCK_SIZE + (ty * BLOCK_SIZE + tx);
}

__device__ __forceinline__ int deviceVid(int x, int y, int z, int3 resolution) {
	return devicePid(x, y, resolution) + z * resolution.x * resolution.y;
}

extern "C"
void cudaInitVolume(int resolutionX, int resolutionY, int resolutionZ, float sizeX, float sizeY, float sizeZ, float centerX, float centerY, float centerZ) {
	resolution.x = resolutionX;
	resolution.y = resolutionY;
	resolution.z = resolutionZ;
	size.x = sizeX;
	size.y = sizeY;
	size.z = sizeZ;
	center.x = centerX;
	center.y = centerY;
	center.z = centerZ;
	hipMalloc(&volume_device, resolution.x * resolution.y * resolution.z * sizeof(float));
	hipMalloc(&volume_color_device, resolution.x * resolution.y * resolution.z * 4 * sizeof(UINT8));
	hipMalloc(&depth_device, H * W * sizeof(float));
	hipMalloc(&color_device, H * W * 4 * sizeof(UINT8));
	hipMalloc(&transformation_device, 16 * sizeof(float));
	hipMalloc(&count_device, resolution.x * resolution.y * sizeof(int));
	count_host = new int[resolution.x * resolution.y];
	block = dim3(BLOCK_SIZE, BLOCK_SIZE);
	grid = dim3((resolution.x + BLOCK_SIZE - 1) / BLOCK_SIZE, (resolution.y + BLOCK_SIZE - 1) / BLOCK_SIZE);
}

extern "C"
void cudaReleaseVolume() {
	hipFree(volume_device);
	hipFree(depth_device);
	hipFree(color_device);
	hipFree(transformation_device);
	hipFree(count_device);
	delete[] count_host;
}

__global__ void kernelIntegrateDepth(float* volume, UINT8* volume_color, UINT16* depthData, UINT8* colorData, float* transformation, int3 resolution, float3 size, float3 center) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	__shared__ float trans_shared[16];
	if (threadIdx.y == 0) {
		trans_shared[threadIdx.x] = transformation[threadIdx.x];
	}

	if (x >= resolution.x || y >= resolution.y) {
		return;
	}

	float volumeSizeX = size.x / resolution.x;
	float volumeSizeY = size.y / resolution.y;
	float volumeSizeZ = size.z / resolution.z;
	float offsetX = center.x - size.x / 2;
	float offsetY = center.y - size.y / 2;
	float offsetZ = center.z - size.z / 2;
	const int resolutionXY = resolution.x * resolution.y;

	const int W = 512;
	const int H = 424;
	const float FX = 367.347;
	const float FY = -367.347;
	const float CX = 260.118;
	const float CY = 208.079;
	const float TRANC_DIST_M = 3.1 * max(volumeSizeX, max(volumeSizeY, volumeSizeZ));

	float oriX = (x + 0.5) * volumeSizeX + offsetX;
	float oriY = (y + 0.5) * volumeSizeY + offsetY;
	for (int z = 0; z < resolution.z; z++) {
		float oriZ = (z + 0.5) * volumeSizeZ + offsetZ;
		float posX = trans_shared[0 + 0] * oriX + trans_shared[0 + 1] * oriY + trans_shared[0 + 2] * oriZ + trans_shared[12 + 0];
		float posY = trans_shared[4 + 0] * oriX + trans_shared[4 + 1] * oriY + trans_shared[4 + 2] * oriZ + trans_shared[12 + 1];
		float posZ = trans_shared[8 + 0] * oriX + trans_shared[8 + 1] * oriY + trans_shared[8 + 2] * oriZ + trans_shared[12 + 2];

		int cooX = posX * FX / posZ + CX;
		int cooY = posY * FY / posZ + CY;

		float tsdf = -1;
		uchar3 color;
		color.x = color.y = color.z = 0;
		if (posZ > 0 && 0 <= cooX && cooX < W && 0 <= cooY && cooY < H) {
			UINT16 depth = depthData[cooY * W + cooX];
			
			if (depth != 0) {
				float xl = (cooX - CX) / FX;
				float yl = (cooY - CY) / FY;
				float sdf = depth * 0.001 - rsqrtf((xl * xl + yl * yl + 1) / (posX * posX + posY * posY + posZ * posZ));

				if (sdf >= -TRANC_DIST_M) {
					tsdf = sdf / TRANC_DIST_M;

					if (tsdf < 1.0) {
						color.x = colorData[((cooY * W + cooX) << 2) + 2];
						color.y = colorData[((cooY * W + cooX) << 2) + 1];
						color.z = colorData[((cooY * W + cooX) << 2) + 0];
					} else {
						tsdf = 1.0;
					}
				}
			}
		}
		__syncthreads();
		int id = deviceVid(x, y, z, resolution);
		volume[id] = tsdf;
		__syncthreads();
		volume_color[(id << 2) + 0] = color.x;
		volume_color[(id << 2) + 1] = color.y;
		volume_color[(id << 2) + 2] = color.z;
		__syncthreads();
		
	}
}

extern "C"
void cudaIntegrateDepth(UINT16* depth, RGBQUAD* color, float* transformation) {
	hipMemcpy(depth_device, depth, H * W * sizeof(UINT16), hipMemcpyHostToDevice);
	hipMemcpy(color_device, color, H * W * 4 * sizeof(UINT8), hipMemcpyHostToDevice);
	hipMemcpy(transformation_device, transformation, 16 * sizeof(float), hipMemcpyHostToDevice);

	kernelIntegrateDepth << <grid, block >> > (volume_device, volume_color_device, depth_device, color_device, transformation_device, resolution, size, center);
}

__constant__ UINT8 triNumber_device[256] = {0, 1, 1, 2, 1, 2, 2, 3, 1, 2, 2, 3, 2, 3, 3, 2, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 2, 3, 4, 4, 3, 3, 4, 4, 3, 4, 5, 5, 2, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 4, 2, 3, 3, 4, 3, 4, 2, 3, 3, 4, 4, 5, 4, 5, 3, 2, 3, 4, 4, 3, 4, 5, 3, 2, 4, 5, 5, 4, 5, 2, 4, 1, 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 3, 2, 3, 3, 4, 3, 4, 4, 5, 3, 2, 4, 3, 4, 3, 5, 2, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 4, 3, 4, 4, 3, 4, 5, 5, 4, 4, 3, 5, 2, 5, 4, 2, 1, 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 2, 3, 3, 2, 3, 4, 4, 5, 4, 5, 5, 2, 4, 3, 5, 4, 3, 2, 4, 1, 3, 4, 4, 5, 4, 5, 3, 4, 4, 5, 5, 2, 3, 4, 2, 1, 2, 3, 3, 2, 3, 4, 2, 1, 3, 2, 4, 1, 2, 1, 1, 0};
__constant__ INT8 triTable_device[256][16] =
{ { -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 3, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 2, 10, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 8, 3, 2, 10, 8, 10, 9, 8, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 11, 2, 8, 11, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 9, 0, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 11, 2, 1, 9, 11, 9, 8, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 10, 1, 11, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 10, 1, 0, 8, 10, 8, 11, 10, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 9, 0, 3, 11, 9, 11, 10, 9, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 10, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 4, 7, 3, 0, 4, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 2, 10, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 10, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1, -1 },
{ 8, 4, 7, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 11, 4, 7, 11, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 0, 1, 8, 4, 7, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 7, 11, 9, 4, 11, 9, 11, 2, 9, 2, 1, -1, -1, -1, -1 },
{ 3, 10, 1, 3, 11, 10, 7, 8, 4, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 11, 10, 1, 4, 11, 1, 0, 4, 7, 11, 4, -1, -1, -1, -1 },
{ 4, 7, 8, 9, 0, 11, 9, 11, 10, 11, 0, 3, -1, -1, -1, -1 },
{ 4, 7, 11, 4, 11, 9, 9, 11, 10, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 10, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 0, 8, 1, 2, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 2, 10, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 10, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1, -1 },
{ 9, 5, 4, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 11, 2, 0, 8, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 5, 4, 0, 1, 5, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 1, 5, 2, 5, 8, 2, 8, 11, 4, 8, 5, -1, -1, -1, -1 },
{ 10, 3, 11, 10, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 9, 5, 0, 8, 1, 8, 10, 1, 8, 11, 10, -1, -1, -1, -1 },
{ 5, 4, 0, 5, 0, 11, 5, 11, 10, 11, 0, 3, -1, -1, -1, -1 },
{ 5, 4, 8, 5, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 7, 8, 9, 5, 7, 10, 1, 2, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1, -1 },
{ 8, 0, 2, 8, 2, 5, 8, 5, 7, 10, 5, 2, -1, -1, -1, -1 },
{ 2, 10, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 7, 9, 5, 7, 8, 9, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 11, -1, -1, -1, -1 },
{ 2, 3, 11, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1, -1 },
{ 11, 2, 1, 11, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 5, 8, 8, 5, 7, 10, 1, 3, 10, 3, 11, -1, -1, -1, -1 },
{ 5, 7, 0, 5, 0, 9, 7, 11, 0, 1, 0, 10, 11, 10, 0, -1 },
{ 11, 10, 0, 11, 0, 3, 10, 5, 0, 8, 0, 7, 5, 7, 0, -1 },
{ 11, 10, 5, 7, 11, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 3, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 0, 1, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 8, 3, 1, 9, 8, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1, -1 },
{ 2, 3, 11, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 11, 0, 8, 11, 2, 0, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 1, 9, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 10, 6, 1, 9, 2, 9, 11, 2, 9, 8, 11, -1, -1, -1, -1 },
{ 6, 3, 11, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 11, 0, 11, 5, 0, 5, 1, 5, 11, 6, -1, -1, -1, -1 },
{ 3, 11, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1, -1 },
{ 6, 5, 9, 6, 9, 11, 11, 9, 8, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 10, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 3, 0, 4, 7, 3, 6, 5, 10, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 9, 0, 5, 10, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1, -1 },
{ 6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1, -1 },
{ 8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1, -1 },
{ 7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9, -1 },
{ 3, 11, 2, 7, 8, 4, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 10, 6, 4, 7, 2, 4, 2, 0, 2, 7, 11, -1, -1, -1, -1 },
{ 0, 1, 9, 4, 7, 8, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1 },
{ 9, 2, 1, 9, 11, 2, 9, 4, 11, 7, 11, 4, 5, 10, 6, -1 },
{ 8, 4, 7, 3, 11, 5, 3, 5, 1, 5, 11, 6, -1, -1, -1, -1 },
{ 5, 1, 11, 5, 11, 6, 1, 0, 11, 7, 11, 4, 0, 4, 11, -1 },
{ 0, 5, 9, 0, 6, 5, 0, 3, 6, 11, 6, 3, 8, 4, 7, -1 },
{ 6, 5, 9, 6, 9, 11, 4, 7, 9, 7, 11, 9, -1, -1, -1, -1 },
{ 10, 4, 9, 6, 4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 10, 6, 4, 9, 10, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 0, 1, 10, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 10, -1, -1, -1, -1 },
{ 1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1, -1 },
{ 0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 4, 9, 10, 6, 4, 11, 2, 3, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 2, 2, 8, 11, 4, 9, 10, 4, 10, 6, -1, -1, -1, -1 },
{ 3, 11, 2, 0, 1, 6, 0, 6, 4, 6, 1, 10, -1, -1, -1, -1 },
{ 6, 4, 1, 6, 1, 10, 4, 8, 1, 2, 1, 11, 8, 11, 1, -1 },
{ 9, 6, 4, 9, 3, 6, 9, 1, 3, 11, 6, 3, -1, -1, -1, -1 },
{ 8, 11, 1, 8, 1, 0, 11, 6, 1, 9, 1, 4, 6, 4, 1, -1 },
{ 3, 11, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1, -1 },
{ 6, 4, 8, 11, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 7, 10, 6, 7, 8, 10, 8, 9, 10, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 7, 3, 0, 10, 7, 0, 9, 10, 6, 7, 10, -1, -1, -1, -1 },
{ 10, 6, 7, 1, 10, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1, -1 },
{ 10, 6, 7, 10, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1, -1 },
{ 2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9, -1 },
{ 7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1, -1 },
{ 7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 3, 11, 10, 6, 8, 10, 8, 9, 8, 6, 7, -1, -1, -1, -1 },
{ 2, 0, 7, 2, 7, 11, 0, 9, 7, 6, 7, 10, 9, 10, 7, -1 },
{ 1, 8, 0, 1, 7, 8, 1, 10, 7, 6, 7, 10, 2, 3, 11, -1 },
{ 11, 2, 1, 11, 1, 7, 10, 6, 1, 6, 7, 1, -1, -1, -1, -1 },
{ 8, 9, 6, 8, 6, 7, 9, 1, 6, 11, 6, 3, 1, 3, 6, -1 },
{ 0, 9, 1, 11, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 7, 8, 0, 7, 0, 6, 3, 11, 0, 11, 6, 0, -1, -1, -1, -1 },
{ 7, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 0, 8, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 1, 9, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 1, 9, 8, 3, 1, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 1, 2, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 10, 3, 0, 8, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 9, 0, 2, 10, 9, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 6, 11, 7, 2, 10, 3, 10, 8, 3, 10, 9, 8, -1, -1, -1, -1 },
{ 7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1, -1 },
{ 10, 7, 6, 10, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 7, 6, 1, 7, 10, 1, 8, 7, 1, 0, 8, -1, -1, -1, -1 },
{ 0, 3, 7, 0, 7, 10, 0, 10, 9, 6, 10, 7, -1, -1, -1, -1 },
{ 7, 6, 10, 7, 10, 8, 8, 10, 9, -1, -1, -1, -1, -1, -1, -1 },
{ 6, 8, 4, 11, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 6, 11, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 6, 11, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 4, 6, 9, 6, 3, 9, 3, 1, 11, 3, 6, -1, -1, -1, -1 },
{ 6, 8, 4, 6, 11, 8, 2, 10, 1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 10, 3, 0, 11, 0, 6, 11, 0, 4, 6, -1, -1, -1, -1 },
{ 4, 11, 8, 4, 6, 11, 0, 2, 9, 2, 10, 9, -1, -1, -1, -1 },
{ 10, 9, 3, 10, 3, 2, 9, 4, 3, 11, 3, 6, 4, 6, 3, -1 },
{ 8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1, -1 },
{ 1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 10, 1, -1, -1, -1, -1 },
{ 10, 1, 0, 10, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 6, 3, 4, 3, 8, 6, 10, 3, 0, 3, 9, 10, 9, 3, -1 },
{ 10, 9, 4, 6, 10, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 9, 5, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 3, 4, 9, 5, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 0, 1, 5, 4, 0, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 11, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1, -1 },
{ 9, 5, 4, 10, 1, 2, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 6, 11, 7, 1, 2, 10, 0, 8, 3, 4, 9, 5, -1, -1, -1, -1 },
{ 7, 6, 11, 5, 4, 10, 4, 2, 10, 4, 0, 2, -1, -1, -1, -1 },
{ 3, 4, 8, 3, 5, 4, 3, 2, 5, 10, 5, 2, 11, 7, 6, -1 },
{ 7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1, -1 },
{ 3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1, -1 },
{ 6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8, -1 },
{ 9, 5, 4, 10, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1, -1 },
{ 1, 6, 10, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4, -1 },
{ 4, 0, 10, 4, 10, 5, 0, 3, 10, 6, 10, 7, 3, 7, 10, -1 },
{ 7, 6, 10, 7, 10, 8, 5, 4, 10, 4, 8, 10, -1, -1, -1, -1 },
{ 6, 9, 5, 6, 11, 9, 11, 8, 9, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 6, 11, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1, -1 },
{ 0, 11, 8, 0, 5, 11, 0, 1, 5, 5, 6, 11, -1, -1, -1, -1 },
{ 6, 11, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 10, 9, 5, 11, 9, 11, 8, 11, 5, 6, -1, -1, -1, -1 },
{ 0, 11, 3, 0, 6, 11, 0, 9, 6, 5, 6, 9, 1, 2, 10, -1 },
{ 11, 8, 5, 11, 5, 6, 8, 0, 5, 10, 5, 2, 0, 2, 5, -1 },
{ 6, 11, 3, 6, 3, 5, 2, 10, 3, 10, 5, 3, -1, -1, -1, -1 },
{ 5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1, -1 },
{ 9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8, -1 },
{ 1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 3, 6, 1, 6, 10, 3, 8, 6, 5, 6, 9, 8, 9, 6, -1 },
{ 10, 1, 0, 10, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1, -1 },
{ 0, 3, 8, 5, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 11, 5, 10, 7, 5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 11, 5, 10, 11, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 11, 7, 5, 10, 11, 1, 9, 0, -1, -1, -1, -1, -1, -1, -1 },
{ 10, 7, 5, 10, 11, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1, -1 },
{ 11, 1, 2, 11, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 11, -1, -1, -1, -1 },
{ 9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 11, 7, -1, -1, -1, -1 },
{ 7, 5, 2, 7, 2, 11, 5, 9, 2, 3, 2, 8, 9, 8, 2, -1 },
{ 2, 5, 10, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 2, 0, 8, 5, 2, 8, 7, 5, 10, 2, 5, -1, -1, -1, -1 },
{ 9, 0, 1, 5, 10, 3, 5, 3, 7, 3, 10, 2, -1, -1, -1, -1 },
{ 9, 8, 2, 9, 2, 1, 8, 7, 2, 10, 2, 5, 7, 5, 2, -1 },
{ 1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 8, 4, 5, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1 },
{ 5, 0, 4, 5, 11, 0, 5, 10, 11, 11, 3, 0, -1, -1, -1, -1 },
{ 0, 1, 9, 8, 4, 10, 8, 10, 11, 10, 4, 5, -1, -1, -1, -1 },
{ 10, 11, 4, 10, 4, 5, 11, 3, 4, 9, 4, 1, 3, 1, 4, -1 },
{ 2, 5, 1, 2, 8, 5, 2, 11, 8, 4, 5, 8, -1, -1, -1, -1 },
{ 0, 4, 11, 0, 11, 3, 4, 5, 11, 2, 11, 1, 5, 1, 11, -1 },
{ 0, 2, 5, 0, 5, 9, 2, 11, 5, 4, 5, 8, 11, 8, 5, -1 },
{ 9, 4, 5, 2, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 5, 10, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1, -1 },
{ 5, 10, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 10, 2, 3, 5, 10, 3, 8, 5, 4, 5, 8, 0, 1, 9, -1 },
{ 5, 10, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1, -1 },
{ 8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1, -1 },
{ 9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 11, 7, 4, 9, 11, 9, 10, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 8, 3, 4, 9, 7, 9, 11, 7, 9, 10, 11, -1, -1, -1, -1 },
{ 1, 10, 11, 1, 11, 4, 1, 4, 0, 7, 4, 11, -1, -1, -1, -1 },
{ 3, 1, 4, 3, 4, 8, 1, 10, 4, 7, 4, 11, 10, 11, 4, -1 },
{ 4, 11, 7, 9, 11, 4, 9, 2, 11, 9, 1, 2, -1, -1, -1, -1 },
{ 9, 7, 4, 9, 11, 7, 9, 1, 11, 2, 11, 1, 0, 8, 3, -1 },
{ 11, 7, 4, 11, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1, -1 },
{ 11, 7, 4, 11, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1, -1 },
{ 2, 9, 10, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1, -1 },
{ 9, 10, 7, 9, 7, 4, 10, 2, 7, 8, 7, 0, 2, 0, 7, -1 },
{ 3, 7, 10, 3, 10, 2, 7, 4, 10, 1, 10, 0, 4, 0, 10, -1 },
{ 1, 10, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1, -1 },
{ 4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 0, 9, 3, 9, 11, 11, 9, 10, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 1, 10, 0, 10, 8, 8, 10, 11, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 1, 10, 11, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 2, 11, 1, 11, 9, 9, 11, 8, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 0, 9, 3, 9, 11, 1, 2, 9, 2, 11, 9, -1, -1, -1, -1 },
{ 0, 2, 11, 8, 0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 3, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 3, 8, 2, 8, 10, 10, 8, 9, -1, -1, -1, -1, -1, -1, -1 },
{ 9, 10, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 2, 3, 8, 2, 8, 10, 0, 1, 8, 1, 10, 8, -1, -1, -1, -1 },
{ 1, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ 0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 },
{ -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1 } };

__device__ __forceinline__ UINT16 deviceGetCubeIndex(float* volume, int x, int y, int z, int3 resolution) {
	if (volume[deviceVid(x + 0, y + 0, z + 0, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 1, y + 0, z + 0, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 0, y + 1, z + 0, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 1, y + 1, z + 0, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 0, y + 0, z + 1, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 1, y + 0, z + 1, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 0, y + 1, z + 1, resolution)] == -1) return 0;
	if (volume[deviceVid(x + 1, y + 1, z + 1, resolution)] == -1) return 0;
	int index = 0;
	if (volume[deviceVid(x + 0, y + 0, z + 0, resolution)] < 0) index |= 1;
	if (volume[deviceVid(x + 1, y + 0, z + 0, resolution)] < 0) index |= 2;
	if (volume[deviceVid(x + 0, y + 1, z + 0, resolution)] < 0) index |= 8;
	if (volume[deviceVid(x + 1, y + 1, z + 0, resolution)] < 0) index |= 4;
	if (volume[deviceVid(x + 0, y + 0, z + 1, resolution)] < 0) index |= 16;
	if (volume[deviceVid(x + 1, y + 0, z + 1, resolution)] < 0) index |= 32;
	if (volume[deviceVid(x + 0, y + 1, z + 1, resolution)] < 0) index |= 128;
	if (volume[deviceVid(x + 1, y + 1, z + 1, resolution)] < 0) index |= 64;
	return index;
}

__global__ void kernelMarchingCubesCount(float* volume, int* count, int3 resolution) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x + 1 >= resolution.x || y + 1 >= resolution.y) {
		if (x == resolution.x - 1 || y == resolution.y - 1) {
			count[devicePid(x, y, resolution)] = 0;
		}
		return;
	}

	int cnt = 0;
	for (int z = 0; z + 1 < resolution.z; z++) {
		int index = deviceGetCubeIndex(volume, x, y, z, resolution);
		cnt += triNumber_device[index];
	}

	count[devicePid(x, y, resolution)] = cnt;
}

__device__ __forceinline__ void deviceCalnEdgePoint(float* volume, UINT8* volume_color, int x1, int y1, int z1, int x2, int y2, int z2, float3& pos, uchar3& color, int3 resolution, float3 size, float3 center) {
	const int resolutionXY = resolution.x * resolution.y;
	int id1 = deviceVid(x1, y1, z1, resolution);
	int id2 = deviceVid(x2, y2, z2, resolution);
	float v1 = volume[id1];
	float v2 = volume[id2];
	if ((v1 < 0) ^ (v2 < 0)) {
		float k =  v1 / (v1 - v2);
		float volumeSizeX = size.x / resolution.x;
		float volumeSizeY = size.y / resolution.y;
		float volumeSizeZ = size.z / resolution.z;
		float offsetX = center.x - size.x / 2;
		float offsetY = center.y - size.y / 2;
		float offsetZ = center.z - size.z / 2;
		pos.x = ((1 - k) * x1 + k * x2 - 0.5) * volumeSizeX + offsetX;
		pos.y = ((1 - k) * y1 + k * y2 - 0.5) * volumeSizeY + offsetY;
		pos.z = ((1 - k) * z1 + k * z2 - 0.5) * volumeSizeZ + offsetZ;
		color.x = (UINT8)min((1 - k) * volume_color[(id1 << 2) + 0] + k * volume_color[(id2 << 2) + 0], 255.0);
		color.y = (UINT8)min((1 - k) * volume_color[(id1 << 2) + 1] + k * volume_color[(id2 << 2) + 1], 255.0);
		color.z = (UINT8)min((1 - k) * volume_color[(id1 << 2) + 2] + k * volume_color[(id2 << 2) + 2], 255.0);
	}
}

__global__ void kernelMarchingCubes(float* volume, UINT8* volume_color, int* count, float* tris, UINT8* tris_color, int3 resolution, float3 size, float3 center) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x + 1 >= resolution.x || y + 1 >= resolution.y) {
		return;
	}

	float3 pos[12];
	uchar3 color[12];

	int tid = count[devicePid(x, y, resolution)];
	for (int z = 0; z + 1 < resolution.z; z++) {
		int index = deviceGetCubeIndex(volume, x, y, z, resolution);

		deviceCalnEdgePoint(volume, volume_color, x + 0, y + 0, z + 0, x + 1, y + 0, z + 0, pos[0], color[0], resolution, size, center);
		deviceCalnEdgePoint(volume, volume_color, x + 1, y + 0, z + 0, x + 1, y + 1, z + 0, pos[1], color[1], resolution, size, center);
		deviceCalnEdgePoint(volume, volume_color, x + 1, y + 1, z + 0, x + 0, y + 1, z + 0, pos[2], color[2], resolution, size, center);
		deviceCalnEdgePoint(volume, volume_color, x + 0, y + 1, z + 0, x + 0, y + 0, z + 0, pos[3], color[3], resolution, size, center);
		
		deviceCalnEdgePoint(volume, volume_color, x + 0, y + 0, z + 1, x + 1, y + 0, z + 1, pos[4], color[4], resolution, size, center);
		deviceCalnEdgePoint(volume, volume_color, x + 1, y + 0, z + 1, x + 1, y + 1, z + 1, pos[5], color[5], resolution, size, center);
		deviceCalnEdgePoint(volume, volume_color, x + 1, y + 1, z + 1, x + 0, y + 1, z + 1, pos[6], color[6], resolution, size, center);
		deviceCalnEdgePoint(volume, volume_color, x + 0, y + 1, z + 1, x + 0, y + 0, z + 1, pos[7], color[7], resolution, size, center);

		deviceCalnEdgePoint(volume, volume_color, x + 0, y + 0, z + 0, x + 0, y + 0, z + 1, pos[8], color[8], resolution, size, center);
		deviceCalnEdgePoint(volume, volume_color, x + 1, y + 0, z + 0, x + 1, y + 0, z + 1, pos[9], color[9], resolution, size, center);
		deviceCalnEdgePoint(volume, volume_color, x + 1, y + 1, z + 0, x + 1, y + 1, z + 1, pos[10], color[10], resolution, size, center);
		deviceCalnEdgePoint(volume, volume_color, x + 0, y + 1, z + 0, x + 0, y + 1, z + 1, pos[11], color[11], resolution, size, center);

		for (int i = 0; i < 5; i++) {
			if (triTable_device[index][i * 3] != -1) {
				for (int j = 0; j < 3; j++) {
					int edgeId = triTable_device[index][i * 3 + j];
					tris[tid * 9 + j * 3 + 0] = pos[edgeId].x;
					tris[tid * 9 + j * 3 + 1] = pos[edgeId].y;
					tris[tid * 9 + j * 3 + 2] = pos[edgeId].z;
					tris_color[tid * 9 + j * 3 + 0] = color[edgeId].x;
					tris_color[tid * 9 + j * 3 + 1] = color[edgeId].y;
					tris_color[tid * 9 + j * 3 + 2] = color[edgeId].z;
				}
				tid++;
			} else {
				break;
			}
		}
	}
}

int cudaCountAccumulation() {
	hipMemcpy(count_host, count_device, resolution.x * resolution.y * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 1; i < resolution.x * resolution.y; i++) {
		count_host[i] += count_host[i - 1];
	}
	for (int i = resolution.x * resolution.y - 1; i >= 1; i--) {
		count_host[i] = count_host[i - 1];
	}
	count_host[0] = 0;
	int tris_size = count_host[resolution.x * resolution.y - 1];
	hipMemcpy(count_device, count_host, resolution.x * resolution.y * sizeof(int), hipMemcpyHostToDevice);
	return tris_size;
}

extern "C"
void cudaCalculateMesh(float*& tris, UINT8*& tris_color, int& tri_size) {
	Timer timer;

	kernelMarchingCubesCount << <grid, block >> > (volume_device, count_device, resolution);
	hipDeviceSynchronize();

	timer.outputTime();

	tri_size = cudaCountAccumulation();

	float* tris_device;
	UINT8* tris_color_device;
	hipMalloc(&tris_device, tri_size * 9 * sizeof(float));
	hipMalloc(&tris_color_device, tri_size * 9 * sizeof(UINT8));

	kernelMarchingCubes << <grid, block >> > (volume_device, volume_color_device, count_device, tris_device, tris_color_device, resolution, size, center);

	tris = new float[tri_size * 9];
	tris_color = new UINT8[tri_size * 9];
	hipMemcpy(tris, tris_device, tri_size * 9 * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(tris_color, tris_color_device, tri_size * 9 * sizeof(UINT8), hipMemcpyDeviceToHost);
	
	hipFree(tris_device);
	hipFree(tris_color_device);
}
