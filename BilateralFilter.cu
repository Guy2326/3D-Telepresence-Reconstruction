#include "hip/hip_runtime.h"
#include "CudaHandleError.h"
#include "Parameters.h"
#include <Windows.h>
#include <math.h>

#define BLOCK_SIZE 16

__global__ void kernelBilateralFiltering(UINT16* depth, UINT16* output, int H, int W) {
	const int RADIUS = 9;
	const float SIGMA_D = 3;
	const float SIGMA_I = 10;
	const float SIGMA_D_C = 2 * SIGMA_D * SIGMA_D;
	const float SIGMA_I_C = 2 * SIGMA_I * SIGMA_I;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < W && y < H) {
		float sum = 0;
		float val = 0;

		for (int dx = -RADIUS; dx <= RADIUS; dx++) {
			for (int dy = -RADIUS; dy <= RADIUS; dy++) {
				int nx = x + dx;
				int ny = y + dy;

				if (0 <= nx && nx < W && 0 <= ny && ny < H) {
					int distD2 = (dx * dx + dy * dy);
					int distI2 = (depth[y * W + x] - depth[ny * W + nx]);
					distI2 = distI2 * distI2;
					float v = exp(-distD2 / SIGMA_D_C - distI2 / SIGMA_I_C);
					val += v;
					sum += v * depth[ny * W + nx];
				}
			}
		}

		if (val > 0) {
			output[y * W + x] = (UINT16)(sum / val);
		}
	}
}

extern "C"
void cudaBilateralFiltering(UINT16* depth) {
	static const int H = DEPTH_H;
	static const int W = DEPTH_W;
	static const int n = H * W;

	UINT16* d_depth;
	UINT16* d_output;
	HANDLE_ERROR(hipMalloc(&d_depth, n * sizeof(UINT16)));
	HANDLE_ERROR(hipMalloc(&d_output, n * sizeof(UINT16)));

	HANDLE_ERROR(hipMemcpy(d_depth, depth, n * sizeof(UINT16), hipMemcpyHostToDevice));

	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blockNum((W + BLOCK_SIZE - 1) / BLOCK_SIZE, (H + BLOCK_SIZE - 1) / BLOCK_SIZE);
	kernelBilateralFiltering << <blockNum, blockSize >> > (d_depth, d_output, H, W);
	HANDLE_ERROR(hipGetLastError());

	HANDLE_ERROR(hipMemcpy(depth, d_output, n * sizeof(UINT16), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipFree(d_depth));
	HANDLE_ERROR(hipFree(d_output));
}
